#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <stdio.h>

__global__ void nbody_kernel(int n, double4 *posData, double4 *velData, int steps) {

  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while(steps--) {
    // position and velocity (last frame)
    double4 pos = posData[index];
    double4 vel = velData[index];

    double4 r;
    double4 acc;

    for (int i = 0; i < n; i++) {

      r = posData[i];
      r.x = r.x - pos.x;
      r.y = r.y - pos.y;
      r.z = r.z - pos.z;

      double distSqr = r.x * r.x + r.y * r.y + r.z * r.z + 0.1;
      double dist = std::sqrt(distSqr);
      double distCube = dist * dist * dist;
      double s = r.w / distCube;

      acc.x = acc.x + r.x * s;
      acc.y = acc.y + r.y * s;
      acc.z = acc.z + r.z * s;

      // No tengo idea de por que, pero este print a veces evita que la simulacion explote
      printf("");


    }

    vel.x = vel.x + acc.x;
    vel.y = vel.y + acc.y;
    vel.z = vel.z + acc.z;

    pos.x = pos.x + vel.x;
    pos.y = pos.y + vel.y;
    pos.z = pos.z + vel.z;

    __syncthreads();
    __threadfence_system();

    posData[index] = pos;
    velData[index] = vel;

    //printf("Particula %d: (%f,%f,%f)\n", index, vel.x, vel.y, vel.z);

  }

};

__device__ float3 batch_calculation(double4 pos, float3 acc, double4* data, int bsize) {
  
  double4 r;
  for (int i = 0; i < bsize; i++) {
    r = data[i];
    r.x -= pos.x;
    r.y -= pos.y;
    r.z -= pos.z;

    double distSqr = r.x * r.x + r.y * r.y + r.z * r.z + 0.1;
    double dist = std::sqrt(distSqr);
    double distCube = dist * dist * dist;
    double s = r.w / distCube;

    acc.x += r.x * s;
    acc.y += r.y * s;
    acc.z += r.z * s;

    // No tengo idea de por que, pero este print evita que la simulacion explote
      printf("");
  }
  return acc;
}

extern __shared__ double4 batchData[];
__global__ void nbody_kernel_shared(int n, double4 *posData, double4 *velData, int steps, int bsize, int bnum) {

  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  while(steps--) {
    // position and velocity (last frame)
    double4 pos = posData[index];
    double4 vel = velData[index];
    float3 acc;

    for (int i = 0; i < bnum; i++) {

      batchData[threadIdx.x] = posData[threadIdx.x + i * blockDim.x];

      __syncthreads();

      acc = batch_calculation(pos, acc, batchData, bsize);

      __syncthreads();
    }

    vel.x += acc.x;
    vel.y += acc.y;
    vel.z += acc.z;

    pos.x += vel.x;
    pos.y += vel.y;
    pos.z += vel.z;

    __syncthreads();
    __threadfence_system();

    posData[index] = pos;
    velData[index] = vel;

    __syncthreads();
  }

};

__global__ void nbody_kernel_2D(int n, double4 *posData, double4 *velData, int steps, int nx) {

  unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int index_y = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int index = index_x + index_y*nx;

  //printf("Index_X: %d, Index_Y: %d, Index: %d\n", index_x, index_y, index);

  while(steps--) {
    // position and velocity (last frame)
    double4 pos = posData[index];
    double4 vel = velData[index];

    double4 r;
    double4 acc;

    for (int i = 0; i < n; i++) {

      r = posData[i];
      r.x = r.x - pos.x;
      r.y = r.y - pos.y;
      r.z = r.z - pos.z;

      double distSqr = r.x * r.x + r.y * r.y + r.z * r.z + 0.1;
      double dist = std::sqrt(distSqr);
      double distCube = dist * dist * dist;
      double s = r.w / distCube;

      acc.x = acc.x + r.x * s;
      acc.y = acc.y + r.y * s;
      acc.z = acc.z + r.z * s;

      // No tengo idea de por que, pero este print evita que la simulacion explote
      printf("");


    }

    vel.x = vel.x + acc.x;
    vel.y = vel.y + acc.y;
    vel.z = vel.z + acc.z;

    pos.x = pos.x + vel.x;
    pos.y = pos.y + vel.y;
    pos.z = pos.z + vel.z;

    __syncthreads();
    __threadfence_system();

    posData[index] = pos;
    velData[index] = vel;
  }

};

__global__ void nbody_kernel_shared_2D(int n, double4 *posData, double4 *velData, int steps, int bsize, int bnumx, int bnumy) {

  unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int index_y = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int index = index_x + index_y*bsize*bnumx;
  unsigned int thread_index = threadIdx.x + threadIdx.y*bsize;

  while(steps--) {
    // position and velocity (last frame)
    double4 pos = posData[index];
    double4 vel = velData[index];
    float3 acc;

    for (int i = 0; i < bnumx; i++) {
      for (int j = 0; j < bnumy; j++) {
        printf("%d\n", thread_index + i * blockDim.x + j * blockDim.y);
        batchData[thread_index] = posData[thread_index + i * blockDim.x + j * blockDim.y];

        __syncthreads();

        acc = batch_calculation(pos, acc, batchData, bsize);

        __syncthreads();
      } 
    }
      


    vel.x += acc.x;
    vel.y += acc.y;
    vel.z += acc.z;

    pos.x += vel.x;
    pos.y += vel.y;
    pos.z += vel.z;

    __syncthreads();
    __threadfence_system();

    posData[index] = pos;
    velData[index] = vel;

    __syncthreads();
  }

};