#include "hip/hip_runtime.h"
#include <chrono>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include "kernel.cuh"

struct Times {
  long create_data;
  long copy_to_host;
  long execution;
  long copy_to_device;
  inline long total() {
    return create_data + copy_to_host + execution + copy_to_device; 
  }
};

Times t;

bool simulate(int N, int Steps, int blockSize, int gridSize) {
  using std::chrono::microseconds;
  std::size_t size = sizeof(float4) * N * 2;
  std::vector<float4> data(2*N);

  // Create the memory buffers
  float4 *dataDev;
  hipMalloc(&dataDev, size);

  // Assign values to host variables
  auto t_start = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < N; i++) {
    data[2*i].x = std::rand() % 1000;
    data[2*i].y = std::rand() % 1000;
    data[2*i].z = std::rand() % 1000;
    data[2*i].w = std::rand() % 25000 + 50000;
    data[2*i + 1] = {0,0,0,0};
  }
  auto t_end = std::chrono::high_resolution_clock::now();
  t.create_data =
      std::chrono::duration_cast<microseconds>(t_end - t_start).count();

  std::cout << "INITIAL: " << std::endl;
  for (int i = 0; i < N; i++)
    std::cout << " Particula " << i << ": (" << data[2*i].x << ", " << data[2*i].y << ", " << data[2*i].z << ")\n";

  // Copy values from host variables to device
  t_start = std::chrono::high_resolution_clock::now();
  hipMemcpy(dataDev, data.data(), size, hipMemcpyHostToDevice);
  t_end = std::chrono::high_resolution_clock::now();
  t.copy_to_device =
      std::chrono::duration_cast<microseconds>(t_end - t_start).count();


  // Execute the function on the device (using 32 threads here)
  t_start = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < Steps; i++) {
    nbody_kernel<<<blockSize, gridSize>>>(N, dataDev, i);
    hipDeviceSynchronize();
  }
  
  t_end = std::chrono::high_resolution_clock::now();
  t.execution =
      std::chrono::duration_cast<std::chrono::microseconds>(t_end - t_start).count();

  // Copy the output variable from device to host
  t_start = std::chrono::high_resolution_clock::now();
  hipMemcpy(data.data(), dataDev, size, hipMemcpyDeviceToHost);
  t_end = std::chrono::high_resolution_clock::now();
  t.copy_to_host =
      std::chrono::duration_cast<std::chrono::microseconds>(t_end - t_start).count();

  // Print the result
  std::cout << "RESULTS: " << std::endl;
  for (int i = 0; i < N; i++)
    std::cout << " Particula " << i << ": (" << data[2*i].x << ", " << data[2*i].y << ", " << data[2*i].z << ")\n";

  std::cout << "Time to create data: " << t.create_data << " microseconds\n";
  std::cout << "Time to copy data to device: " << t.copy_to_device
            << " microseconds\n";
  std::cout << "Time to execute kernel: " << t.execution << " microseconds\n";
  std::cout << "Time to copy data to host: " << t.copy_to_host
            << " microseconds\n";
  std::cout << "Time to execute the whole program: " << t.total()
            << " microseconds\n";

  hipFree(dataDev);

  return true;

}

int main(int argc, char* argv[]) {
  if (argc != 5) {
    std::cerr << "Uso: " << argv[0] << " <array size> <step_count> <block size> <grid size>"
              << std::endl;
    return 2;
  }
  int n = std::atoi(argv[1]);
  int s = std::atoi(argv[2]);
  int bs = std::atoi(argv[3]);
  int gs = std::atoi(argv[4]);

  if (!simulate(n, s, bs, gs)) {
    std::cerr << "CUDA: Error while executing the simulation" << std::endl;
    return 3;
  }

  // std::ofstream out;
  // out.open(argv[5], std::ios::app | std::ios::out);
  // if (!out.is_open()) {
  //   std::cerr << "Error while opening file: '" << argv[2] << "'" << std::endl;
  //   return 4;
  // }
  // // params
  // out << n << "," << bs << "," << gs << ",";
  // // times
  // out << t.create_data << "," << t.copy_to_device << "," << t.execution << "," << t.copy_to_host << "," << t.total() << "\n";

  // std::cout << "Data written to " << argv[4] << std::endl;
  return 0;
}
